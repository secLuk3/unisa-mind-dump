#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void prodottoScalareCPU(float *a, float *b, float *c, int n);
float sommaCPU(float *a, int n);

__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n);

int main(void)
{
	float *w_h, *w_h2, s_h, *u_h, *v_h; // host data -> CPU
	float *w_d, *u_d, *v_d, s_d; // device data -> GPU
	int N, nBytes, i, sharedDim;
	float tempoGPU, tempoCPU;
	dim3 gridDim, blockDim;

	printf("***\t Prodotto scalare vettori - 2 strategia \t***\n");

	printf("Inserisci il numero degli elementi dei vettori\n");
	scanf("%d", &N);

	blockDim.x = 64; //Configurazione ottimale
	
	// Determinazione esatta del numero di blocchi
	gridDim.x = N/blockDim.x+((N%blockDim.x)==0?0:1);

     // Dimensione dell' array condiviso fra i thread allocato dinamicamente
    sharedDim = blockDim.x * sizeof(float);

	// Inizializzazione variabili
	nBytes = N * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);
	s_h = 0;
  	s_d = 0;

	hipMalloc((void **)&w_d, nBytes);
	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);

	//Inizializzazione dati in modo randomico
	srand((unsigned int)time(0));

	for (i = 0; i < N; i++)
	{
		u_h[i] = rand() % 5 - 2;
		v_h[i] = rand() % 5 - 2;
		
	}

	// cudamemcpy( vettore destinazione, vettore da copiare, n byte da copiare, tipo di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// Azzero il contenuto del vettore w e gli riservo lo spazio indicato
	memset(w_h, 0, gridDim.x * sizeof(float));
	hipMemset(w_d, 0, nBytes);

	// Calcolo del tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	//Invocazione del kernel
	prodottoScalareGPU<<<gridDim, blockDim, sharedDim>>>(u_d, v_d, w_d, N);
	hipMemcpy(w_h, w_d, gridDim.x * sizeof(float), hipMemcpyDeviceToHost);

  	//Unione dei sottorisultati ottenuti del vettore ottenuto 
	//L'array avrà nella posizione iniziale di ogni blocco la somma parziale dei vari blocchi
  	for(i=0; i< gridDim.x; i++)
  	{
    	s_d += w_h[i];
  	}


	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo

	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&tempoGPU, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo GPU = %f\n", tempoGPU);

	//Calcolo tempo esecuzione CPU
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	//Calcolo prodotto scalare seriale su CPU
	prodottoScalareCPU(u_h, v_h, w_h2, N);
	s_h = sommaCPU(w_h2, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tempoCPU, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo CPU = %f\n", tempoCPU);

	//Controllo se il risultato corrisponde
	assert(s_h == s_d);

	//Stampa risultati
	if (N < 20)
	{
		for (i = 0; i < N; i++)
			printf("u_h[%d]=%.2f ", i, u_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("v_h[%d]=%.2f ", i, v_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("w_h2[%d]=%.2f ", i, w_h2[i]);
		printf("\n");
		for (i = 0; i < gridDim.x; i++)
			printf("w_h[%d]=%.2f ", i, w_h[i]);
		printf("\n");
	}

	printf("\n---> Il prodotto scalare calcolato dalla CPU è: %.2f\n", s_h);
  	printf("\n---> Il prodotto scalare calcolato dalla GPU è: %.2f", s_d);

	//Rilascio memoria utilizzata
	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void prodottoScalareCPU(float *a, float *b, float *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];
}

// Seriale
float sommaCPU(float *a, int n)
{
	int i;
	float s;

	for (i = 0; i < n; i++)
	{
		s += a[i];
	}

	return s;
}

//Prodotto Scalare Parallelo con Somma Parallela Ottimizzata
//Parallelo ma con bank conflict. 
// -> !! Thread diversi accedono a elementi diversi dello stesso banco
__global__ void prodottoScalareGPU(float* a, float * b, float* c, int n){
    int passo, distanza, passiTotali = 0;

    extern __shared__ float s[];
    int index=threadIdx.x + blockIdx.x*blockDim.x;
    int id = threadIdx.x;

    if(index < n)
        s[id] = a[index] * b[index];
    __syncthreads();

    // Somma in parallelo  
    passo = blockDim.x;
    while(passo != 1){ /* Shifta di un bit a destra*/
		passo = passo >> 1; //Divisione per due
		passiTotali++; //Calcolo logaritmo numero passi totale
	}

	for(int i=0;i<passiTotali;i++){
	    distanza = 1 << i; //Calcolo potenze di 2^i (di 2)

	    if(id % (distanza * 2) == 0){ //Comunicano i processi 
	        s[id] = s[id] + s[id + distanza];
	    }

	    __syncthreads();
	}

	if(id == 0) c[blockIdx.x] = s[0];
}
