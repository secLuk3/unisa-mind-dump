#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void prodottoScalareCPU(float *a, float *b, float *c, int n);
float sommaCPU(float *a, int n);
__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n);

int main(void)
{
	float *w_h, *w_h2, s_h, *u_h, *v_h; // host data -> CPU
	float *w_d, *u_d, *v_d, s_d;	// device data -> GPU
	int N, nBytes, i;
	float tempoGPU, tempoCPU;
	dim3 gridDim, blockDim;

	printf("***\t PRODOTTO DI DUE VETTORI \t***\n");

	printf("Inserisci il numero degli elementi dei vettori\n");
	scanf("%d", &N);

	blockDim.x = 64; //Configurazione ottimale
	
	// Determinazione esatta del numero di blocchi
	gridDim.x = N/blockDim.x+((N%blockDim.x)==0?0:1);
	printf("Grid dim x: %d", gridDim.x);

	//Inizializzazione variabili
	nBytes = N * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);
	s_h = 0;
	s_d = 0;

	hipMalloc((void **)&w_d, nBytes);
	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);

	//Inizializzazione dati in modo randomico
	srand((unsigned int)time(0));

	for (i = 0; i < N; i++)
	{
		u_h[i] = rand() % 5 - 2;
		v_h[i] = rand() % 5 - 2;
		
	}

	// cudamemcpy( vettore destinazione, vettore da copiare, n byte da copiare, tipo di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// Azzero il contenuto del vettore w e gli riservo lo spazio indicato
	memset(w_h, 0, nBytes);
	hipMemset(w_d, 0, nBytes);

	// Calcolo del tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	//Invocazione del kernel
	prodottoScalareGPU<<<gridDim, blockDim>>>(u_d, v_d, w_d, N);

	//Riporto il valore calcolato dalla GPU alla CPU
	hipMemcpy(w_h, w_d, nBytes, hipMemcpyDeviceToHost);
	s_d = sommaCPU(w_h, N);

	
	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&tempoGPU, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("tempo GPU = %f\n", tempoGPU);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	//Calcolo prodotto scalare seriale su CPU
	prodottoScalareCPU(u_h, v_h, w_h2, N);
	s_h = sommaCPU(w_h2, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tempoCPU, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("tempo CPU = %f\n", tempoCPU);

	// verifica che i risultati di CPU e GPU siano uguali
	// se non stampa nulla, i due vettori sono uguali
	assert(s_d == s_h);

	//Stampa risultati
	if (N < 20)
	{
		for (i = 0; i < N; i++)
			printf("u_h[%d]=%.2f ", i, u_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("v_h[%d]=%.2f ", i, v_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("w_h[%d]=%.2f ", i, w_h[i]);
		printf("\n");
	}

	printf("\n---> Il prodotto scalare calcolato dalla CPU è: %.2f\n", s_h);
  	printf("\n---> Il prodotto scalare calcolato dalla GPU è: %.2f", s_d);

	//Rilascio memoria 
	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void prodottoScalareCPU(float *a, float *b, float *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];
}

// Seriale
float sommaCPU(float *a, int n)
{
	int i;
	float s;

	for (i = 0; i < n; i++)
	{
		s += a[i];
	}

	return s;
}

// Prodotto scalare parallelo ma non troppo
// Non sfrutto la memoria condivisa (non vogliamo questo) ma quella globale

__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		c[index] = a[index] * b[index];
}
