#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void prodottoScalareCPU(float *a, float *b, float *c, int n);
float sommaCPU(float *a, int n);
__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n);

int main(void)
{
	float *w_h, *w_h2, s_h, *u_h, *v_h;  // host data -> CPU
	float *w_d, *u_d, *v_d, s_d; // device data -> GPU
	int N, nBytes, i, sharedDim;
	float tempoGPU, tempoCPU;
	dim3 gridDim, blockDim;

	printf("***\t Prodotto scalare vettori - 3 strategia \t***\n");

	printf("Inserisci il numero degli elementi dei vettori\n");
	scanf("%d", &N);

	blockDim.x = 64; //Configurazione ottimale
	
	// Determinazione esatta del numero di blocchi
	gridDim.x = N/blockDim.x+((N%blockDim.x)==0?0:1);

    // Dimensione dell' array condiviso fra i thread allocato dinamicamente
    sharedDim = blockDim.x * sizeof(float);

	// Inzializzazione variabili
	nBytes = N * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);
	s_h = 0;
	s_d = 0;

	hipMalloc((void **)&w_d, nBytes);
	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);

	//Inizializzazione dati in modo randomico
	srand((unsigned int)time(0));

	for (i = 0; i < N; i++)
	{
		u_h[i] = rand() % 5 - 2;
		v_h[i] = rand() % 5 - 2;
		
	}

	// Ricorda cudamemcpy( vettore destinazione, vettore da copiare, byte da copiare, verso di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// Azzero il contenuto del vettore w e gli riservo lo spazio indicato
	memset(w_h, 0, gridDim.x * sizeof(float));
	hipMemset(w_d, 0, nBytes);

	// Calcolo del tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start); //Start timer

	//Invocazione del kernel
	prodottoScalareGPU<<<gridDim, blockDim, sharedDim>>>(u_d, v_d, w_d, N); //Con allocazione sharedMemory dinamica
	hipMemcpy(w_h, w_d, gridDim.x * sizeof(float), hipMemcpyDeviceToHost); //Passaggio dei dati ottenuti all'host

  	//Unione dei sottorisultati ottenuti del vettore ottenuto 
	//L'array avrà nella posizione iniziale di ogni blocco la somma parziale dei vari blocchi
  	for(i=0; i< gridDim.x; i++)
  	{
    	s_d += w_h[i];
  	}

	hipEventRecord(stop); //Stop timer
	hipEventSynchronize(stop); //Sincornizzazione del timer -> Assicura che tutti siano arrivati all'evento stop prima di registrare il tempo finale

	
	//Calcolo del tempo tra i due eventi -> in millisecondi
	hipEventElapsedTime(&tempoGPU, start, stop);
	hipEventDestroy(start); 
	hipEventDestroy(stop); 
	
	printf("tempo GPU = %f\n", tempoGPU);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// Calcolo prodotto scalare seriale su CPU
	prodottoScalareCPU(u_h, v_h, w_h2, N);
	s_h = sommaCPU(w_h2, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tempoCPU, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo CPU = %f\n", tempoCPU);

	//Controllo se il risultato corrisponde
	assert(s_h == s_d);

	if (N < 20)
	{
		for (i = 0; i < N; i++)
			printf("u_h[%d]=%.2f ", i, u_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("v_h[%d]=%.2f ", i, v_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("w_h2[%d]=%.2f ", i, w_h2[i]);
		printf("\n");

		for (i = 0; i < gridDim.x; i++)
			printf("w_h[%d]=%.2f ", i, w_h[i]);
		printf("\n");
	}

	printf("\n---> Il prodotto scalare calcolato dalla CPU è: %.2f\n", s_h);
  	printf("\n---> Il prodotto scalare calcolato dalla GPU è: %.2f", s_d);

	//Rilascio memoria utilzzata
	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void prodottoScalareCPU(float *a, float *b, float *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];
}

// Seriale
float sommaCPU(float *a, int n)
{
	int i;
	float s;

	for (i = 0; i < n; i++)
	{
		s += a[i];
	}

	return s;
}

//Prodotto scalare parallelo
// -> Senza bank conflict poichè thread diversi accedono a elementi diversi di banchi diversi
// Con limitazione:
// - blockDim deve essere potenza di 2
__global__ void prodottoScalareGPU(float* a, float * b, float* c, int n){

	int distanza = blockDim.x / 2;
    extern __shared__ float s[];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = threadIdx.x;

    if(index < n)
        s[id] = a[index] * b[index];
    __syncthreads();

    // Somma in parallelo  
  	while(distanza > 0 ){
		
	    if(id < distanza){ //Lavorano thread con lo stesso id ma di warp diversi
	        s[id] = s[id] + s[id + distanza];
	    }
	    __syncthreads();

		distanza /= 2; //Divido ogni volta la distanza per 2
	}

	if(id == 0) c[blockIdx.x] = s[0];
}
