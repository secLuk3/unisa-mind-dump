#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void prod_mat_v(float w[], float *a, int ROWS, int COLS, float v[]);
void generaMatrice(float *A, int M, int N);
void stampa_matrice(float *A, int M, int N);
void generaVettore(float *v, int N);
void stampaVettore(float *v, int N);

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float* h_A = 0;     // Host matrice A
    float* d_A;         // Device array A
    float* h_b = 0;     // Host array b
    float *d_b;         // Device array b
    float *h_r = 0;     // Host risultato
    float *d_r;         // Device risultato

	int M; //Numero righe
    int N; //Numero colonne
    float *h_v;  //Risultato seriale

    printf("Inserisci numero di righe M della matrice: ");
    scanf("%d", &M);

    printf("Inserisci numero di colonne N della matrice: ");
    scanf("%d", &N);

    h_A = (float *)malloc (M * N * sizeof (*h_A));      // Alloco h_A e lo inizializzo
    if (!h_A) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    
    generaMatrice(h_A, M, N);

    h_b = (float *)malloc (N * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    generaVettore(h_b, N);

    if(M < 10 && N < 10){
        stampa_matrice(h_A, M, N);
        stampaVettore(h_b, N);
    }


    cudaStat = hipMalloc ((void**)&d_A, M * N * sizeof(*h_A));       // Alloco d_a
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&d_b, N*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetMatrix(M, N, sizeof(float), h_A, M, d_A, M);    // Setto h_A su d_A
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Matrice non impostata");
        hipFree (d_A);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(N, sizeof(float), h_b, 1, d_b, 1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Vettore molitplicatore non impostato! ");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    h_r = (float *)malloc(M * sizeof(float));
    if (!h_r) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }


    cudaStat = hipMalloc ((void**)&d_r, M*sizeof(*h_r));       // Alloco d_r
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M, sizeof(float), h_r, 1, d_r, 1);    // Setto h_r su d_r
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Vettore risultato non scaricato!");
        hipFree (d_r);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    
    
     // parte calcolo GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float alfa = 1, beta = 0;
    stat = hipblasSgemv(handle, HIPBLAS_OP_T, M, N, &alfa, d_A, M, d_b, 1, &beta, d_r, 1);
    hipMemcpy(h_r, d_r, M * sizeof(float), hipMemcpyDeviceToHost); //Passo i dati all'host

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo

    
    float tempoGPU;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&tempoGPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\ntempo GPU=%f\n", tempoGPU);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_A);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

     // calcolo su CPU
    h_v = (float *)malloc(N*sizeof(float));
    if(!h_v){
         printf ("Allocazione memoria fallita!!");
        return EXIT_FAILURE;
    }

    float tempoCPU;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

     // calcolo somma seriale
    prod_mat_v(h_v, h_A, M, N, h_b);

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    hipEventElapsedTime(&tempoCPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("\ntempo CPU=%f\n", tempoCPU);


    //Controllo se i risultati sono uguali
    for (int i = 0; i < M ; i++)
	{
       assert(h_v[i] == h_r[i]);	
	}

    
    if(M < 10){
        printf("Risultato del prodotto mat-vet su GPU = \n");
        stampaVettore(h_r, M);

        printf("Risultato del prodotto mat-vet su CPU = \n");
        stampaVettore(h_v, M);
    }


    hipFree (d_A);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    hipFree (d_r);     // Dealloco d_b
    
    hipblasDestroy(handle);  // Distruggo l'handle
    
    free(h_A);      // Dealloco h_a
    free(h_b);      // Dealloco h_b    
    free(h_r);
    free(h_v);

    return EXIT_SUCCESS;
}


/**
 * Funzione che esegue il prodotto matrice vettore
*/
void prod_mat_v(float w[], float *a, int ROWS, int COLS, float v[])
{
    int i, j;

    for (i = 0; i < ROWS; i++)
    {
        w[i] = 0;
        for (j = 0; j < COLS; j++)
        {
            w[i] += a[i * COLS + j] * v[j];
        }
    }
}

/**
* Funzione che esegue la stampa delle matrice 
* 
*/
void stampa_matrice(float *A, int M, int N)
{
    int i, j;
    printf("\nMatrice = \n");
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
        {
            printf("\t%.2f", A[i * M + j]);
        }
        printf("\n");
    }
}

/**
* Funzione che genera un vettore di dimensione N 
* 
*/
void generaVettore(float *v, int N){
    int j;
    printf("\n...Genero vettore di dimensione %d...", N);
    for (j = 0; j < N; j++)
    {
        v[j] = j;
    }

}

/**
* Funzione che stampa un vettore di dimensione N 
* 
*/
void stampaVettore(float *v, int N){
    int j;
    printf("\nVettore = \n");
    for (j = 0; j < N; j++)
    {
          printf("\t%.2f\n", v[j]);
    }
    
}

/**
* Funzione che genera una matrice di dimensione M, N 
* 
*/
void generaMatrice(float *A, int M, int N){
    
    int i,j;

    printf("\n...Genero matrice di dimensione %dx%d...",M, N);
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < N; j++)
        {
            if (j == 0)
                A[i * N + j] = (float) ( rand() % 2 ) - 2;
            else
                A[i * N + j] = (float) ( rand() % 5 ) - 2;
        }
    }

}

    