#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int prodottoScalareCPU(float *a, float *b, float *c, int n);

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float* h_a = 0;     // Host array a
    float* d_a;         // Device array a
    float* h_b = 0;     // Host array b
    float *d_b;         // Device array b
    float risultatoGPU = 0;   // Risultato finale GPU
    float risultatoCPU = 0;  // Risultato finale CPU

	int M; //Dimensioni vettore
    float *h_v; 
	/*
	[3, 10, 20] * [5, 10, 15] = 415
	*/

    printf("Inserisci dimensione dei vettori: ");
    scanf("%d", &M);


    h_a = (float *)malloc (M * sizeof (*h_a));      // Alloco h_a e lo inizializzo
    if (!h_a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    
    h_b = (float *)malloc (M * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    //Generazione elementi vettore a caso
    for (int i=0; i<M; i++) {
        h_a[i] = rand()%5-2;
    	h_b[i] = rand()%5-2;
    }

    cudaStat = hipMalloc ((void**)&d_a, M*sizeof(*h_a));       // Alloco d_a
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&d_b, M*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_a,1,d_a,1);    // Setto h_a su d_a
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_a);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    // parte calcolo GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    stat = hipblasSdot(handle,M,d_a,1,d_b,1,&risultatoGPU);        // Calcolo il prodotto
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_a);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    float timeGPU;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&timeGPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo GPU -->%f\n", timeGPU);
    
   

    h_v = (float *) malloc(M*sizeof(float*));

    // Parte di calcolo CPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    risultatoCPU = prodottoScalareCPU(h_a, h_b, h_v, M);

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    float timeCPU;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&timeCPU, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo CPU -->%f\n", timeCPU);

    assert(risultatoGPU == risultatoCPU);

    printf("Risultato del prodotto su GPU --> %f\n",risultatoGPU);
    printf("Risultato del prodotto su CPU --> %f\n",risultatoCPU);

    hipFree (d_a);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    
    hipblasDestroy(handle);  // Distruggo l'handle
    
    free(h_a);      // Dealloco h_a
    free(h_b);      // Dealloco h_b    
    free(h_v);
    return EXIT_SUCCESS;
}


// Seriale
int prodottoScalareCPU(float *a, float *b, float *c, int n)
{
	int i, result = 0;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];

    for (i = 0; i < n; i++)
		result += c[i];

    return result;

}