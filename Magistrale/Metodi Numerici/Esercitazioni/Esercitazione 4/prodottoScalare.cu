//CIAO
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void prodottoScalareCPU(float *a, float *b, float *c, int n);
float sommaCPU(float *a, int n);

__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n);

int main(void)
{
	float *w_h, *w_h2, s_h, *u_h, *v_h; // host data
	float *w_d, *u_d, *v_d;				// device data
	int N, nBytes, i;
	dim3 gridDim, blockDim;

	printf("***\t PRODOTTO DI DUE VETTORI \t***\n");

	printf("Inserisci il numero degli elementi dei vettori\n");
	scanf("%d", &N);

	printf("Inserisci il numero di thread per blocco\n");
	scanf("%d", &blockDim.x);

	// determinazione esatta del numero di blocchi
	gridDim = N / blockDim.x +
			  ((N % blockDim.x) == 0 ? 0 : 1);

	// izializzazione variabili
	nBytes = N * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);
	s_h = 0;

	hipMalloc((void **)&w_d, nBytes);
	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);

	// inizializzo i dati
	/*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
	srand((unsigned int)time(0));

	for (i = 0; i < N; i++)
	{
		u_h[i] = rand() % 5 - 2;
		v_h[i] = rand() % 5 - 2;
		
	}

	// cudamemcpy( vettore destinazione, vettore da copiare, n byte da copiare, tipo di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// azzeriamo il contenuto del vettore w
	memset(w_h, 0, nBytes);
	hipMemset(w_d, 0, nBytes);

	// inizializzo i contatori per il tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// invocazione del kernel
	prodottoScalareGPU<<<gridDim, blockDim>>>(u_d, v_d, w_d, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo

	float elapsed;
	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo GPU = %f\n", elapsed);

	hipMemcpy(w_h, w_d, nBytes, hipMemcpyDeviceToHost);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	// calcolo prodotto scalare seriale su CPU
	prodottoScalareCPU(u_h, v_h, w_h2, N);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo CPU = %f\n", elapsed);

	// verifica che i risultati di CPU e GPU siano uguali
	// se non stampa nulla, i due vettori sono uguali
	for (i = 0; i < N; i++)
		assert(w_h[i] == w_h2[i]);

	if (N < 20)
	{
		for (i = 0; i < N; i++)
			printf("u_h[%d]=%6.2f ", i, u_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("v_h[%d]=%6.2f ", i, v_h[i]);
		printf("\n");
		for (i = 0; i < N; i++)
			printf("w_h[%d]=%6.2f ", i, w_h[i]);
		printf("\n");
	}

	s_h = sommaCPU(w_h, N);
	printf("Il prodotto è: %6.2f", s_h);

	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void prodottoScalareCPU(float *a, float *b, float *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];
}

// Seriale
float sommaCPU(float *a, int n)
{
	int i;
	float s;

	for (i = 0; i < n; i++)
	{
		s += a[i];
	}

	return s;
}

// Parallelo
__global__ void prodottoScalareGPU(float *a, float *b, float *c, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n)
		c[index] = a[index] * b[index];
}
