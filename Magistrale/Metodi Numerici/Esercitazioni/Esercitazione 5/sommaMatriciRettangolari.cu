#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void sommaMatriciCPU(float *a, float *b, float *c, int n);

__global__ void sommaMatriciGPU(float *a, float *b, float *c, int n);

void stampaMatrice(float *a, int M, int N);

int main(void)
{
	float *w_h, *w_h2, *u_h, *v_h; // host data
	float *w_d, *u_d, *v_d;		   // device data
	int N, M, nBytes, i, dimMatrix;
	dim3 gridDim, blockDim;

	printf("***\t SOMMA DI DUE MATRICI RETTANGOLARI \t***\n");

	printf("Inserisci il numero delle righe matrice M x N\n");
	scanf("%d", &M);

	printf("Inserisci il numero delle colonne della matrice M x N\n");
	scanf("%d", &N);

	dimMatrix = M * N;

	// configurazione kernel
	blockDim.x = 16;
	blockDim.y = 4;
	gridDim.x = M / blockDim.x + ((M % blockDim.x) == 0 ? 0 : 1);
	gridDim.y = N / blockDim.y + ((N % blockDim.y) == 0 ? 0 : 1);


	// izializzazione variabili
	nBytes = dimMatrix * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);

	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);
	hipMalloc((void **)&w_d, nBytes);

	// inizializzo i dati
	/*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
	srand((unsigned int)time(0));

	for (i = 0; i < dimMatrix; i++)
	{
		u_h[i] = rand() % 5 - 2;
		v_h[i] = rand() % 5 - 2;
	}

	// cudamemcpy( vettore destinazione, vettore da copiare, n byte da copiare, tipo di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// azzeriamo il contenuto del vettore w
	memset(w_h, 0, nBytes);
	hipMemset(w_d, 0, nBytes);

	// inizializzo i contatori per il tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// invocazione del kernel
	sommaMatriciGPU<<<gridDim, blockDim>>>(u_d, v_d, w_d, dimMatrix);

	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo

	float elapsed;
	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo GPU = %f\n\n", elapsed);

	hipMemcpy(w_h, w_d, nBytes, hipMemcpyDeviceToHost);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// calcolo somma matrice seriale su CPU
	sommaMatriciCPU(u_h, v_h, w_h2, dimMatrix);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo CPU = %f\n\n", elapsed);


	// verifica che i risultati di CPU e GPU siano uguali
	// se non stampa nulla, i due vettori sono uguali
	for (i = 0; i < dimMatrix; i++)
	{
		assert(w_h[i] == w_h2[i]);
	}

	if (N < 20)
	{
		printf("Matrice U da host\n");
		stampaMatrice(u_h, M, N);

		printf("Matrice V da host\n");
		stampaMatrice(v_h, M, N);

		printf("Matrice somma risultante da device\n");
		stampaMatrice(w_h, M, N);
	}

	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void sommaMatriciCPU(float *a, float *b, float *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] + b[i];
}

// Parallelo
__global__ void sommaMatriciGPU(float *a, float *b, float *c, int n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = j * gridDim.x * blockDim.x + i;

	if (index < n)
		c[index] = a[index] + b[index];
}

// stampa matrice
void stampaMatrice(float *a, int M, int N)
{
	int i, j;

	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%.2f\t", a[i*N+j]);
		}
		printf("\n");
	}
}
