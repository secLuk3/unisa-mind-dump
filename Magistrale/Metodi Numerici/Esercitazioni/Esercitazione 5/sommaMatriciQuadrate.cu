#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

void sommaMatriciCPU(float *a, float *b, float *c, int m, int n);

__global__ void sommaMatriciGPU(float *a, float *b, float *c, int m, int n);

void stampaMatrice(float *a, int M);

int main(void)
{
	float *w_h, *w_h2, *u_h, *v_h; // host data
	float *w_d, *u_d, *v_d;		   // device data
	int M, nBytes, i, j,  dimMatrix;
	dim3 gridDim, blockDim;

	printf("***\t SOMMA DI DUE MATRICI QUADRATE \t***\n");

	printf("Inserisci le dimensioni delle matrici M x M\n");
	scanf("%d", &M);

	dimMatrix = M * M;

	//Configurazione Kernel

	//Dimensione del singolo blocco (1D, 2D o 3D)
	blockDim.x = 32; 
	blockDim.y = 32;

	// determinazione esatta del numero di blocchi
	gridDim.x = M / blockDim.x +
			  ((M % blockDim.x) == 0 ? 0 : 1);
	gridDim.y = M / blockDim.y +
			  ((M % blockDim.y) == 0 ? 0 : 1);

	// izializzazione variabili
	nBytes = dimMatrix * sizeof(float);
	u_h = (float *)malloc(nBytes);
	v_h = (float *)malloc(nBytes);
	w_h = (float *)malloc(nBytes);
	w_h2 = (float *)malloc(nBytes);

	hipMalloc((void **)&w_d, nBytes);
	hipMalloc((void **)&u_d, nBytes);
	hipMalloc((void **)&v_d, nBytes);

	// inizializzo i dati
	/*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
	srand((unsigned int)time(0));

	for (i = 0; i < M; i++)
	{
		for(j=0; j< M; j++){
			u_h[i*M+j] = rand() % 5 - 2;
			v_h[i*M+j] = rand() % 5 - 2;
		}
		
	}

	// cudamemcpy( vettore destinazione, vettore da copiare, n byte da copiare, tipo di copia)
	hipMemcpy(u_d, u_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(v_d, v_h, nBytes, hipMemcpyHostToDevice);

	// azzeriamo il contenuto del vettore w
	memset(w_h, 0, nBytes);
	hipMemset(w_d, 0, nBytes);

	// inizializzo i contatori per il tempo di eseciuzione su GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// invocazione del kernel
	sommaMatriciGPU<<<gridDim, blockDim>>>(u_d, v_d, w_d, M, M);

	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo

	float elapsed;
	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo GPU = %f\n\n", elapsed);

	hipMemcpy(w_h, w_d, nBytes, hipMemcpyDeviceToHost);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	// calcolo prodotto scalare seriale su CPU
	sommaMatriciCPU(u_h, v_h, w_h2, M, M);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("tempo CPU = %f\n", elapsed);

	// verifica che i risultati di CPU e GPU siano uguali
	// se non stampa nulla, i due vettori sono uguali
	for (i = 0; i < M; i++)
	{
		for(j= 0; j < M; j++){
			assert(w_h[i*M+j] == w_h2[i*M+j]);
		}
	}
		

	if (M < 20)
	{
		printf("Matrice U\n");
		stampaMatrice(u_h, M);

		printf("Matrice V\n");
		stampaMatrice(v_h, M);

		printf("Matrice somma risultante\n");
		stampaMatrice(w_h, M);
	}

	free(u_h);
	free(v_h);
	free(w_h);
	free(w_h2);

	hipFree(u_d);
	hipFree(v_d);
	hipFree(w_d);

	return 0;
}

// Seriale
void sommaMatriciCPU(float *a, float *b, float *c, int m, int n)
{
	int i, j;
	for (i = 0; i < m; i++){
		for(j = 0; j < n; j++){
			c[i*n+j] = a[i*n+j] + b[i*n+j];
		}
	}
		
}

// Parallelo
__global__ void sommaMatriciGPU(float *a, float *b, float *c, int m, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int index = j * gridDim.x  *  blockDim.x + i;
	
	if (index < n*m)
		c[index] = a[index] + b[index];
}

//stampa matrice
void stampaMatrice(float *a, int M)
{
	int i, j;

	for (i = 0; i < M; i++)
	{
		for(j = 0; j < M; j++){
			printf("%.2f\t", a[i*M+j]);
		}
		printf("\n");
	}
}
